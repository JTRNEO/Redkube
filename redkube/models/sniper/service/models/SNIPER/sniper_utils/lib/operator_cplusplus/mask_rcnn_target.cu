#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * Copyright (c) 2018 University of Maryland, College Park
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file mask_rcnn_target.cu
 * \brief MaskRcnnTarget Operator
 * \author Mahyar Najibi, Bharat Singh
*/

#include "./mask_rcnn_target-inl.h"
#include "../coco_api/common/maskApi.h"
#include <set>
#include <math.h>
#include <unistd.h>
#include <dmlc/logging.h>
#include <dmlc/parameter.h>
#include <mxnet/operator.h>
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "./operator_common.h"
#include "./mshadow_op.h"
#include <time.h>


namespace mxnet {
namespace op {
namespace mask_utils {
    // Mask Utility Functions
    inline void convertPoly2Mask(const float* roi, const float* poly, const int mask_size, float* mask, float category)
    {
     /* !
     Converts a polygon to a pre-defined mask wrt to an roi
     *****Inputs****
     roi: The RoI bounding box 
     poly: The polygon points the pre-defined format(see below)
     mask_size: The mask size
     *****Outputs****
     overlap: overlap of each box in boxes1 to each box in boxes2
     */
      float w = roi[3] - roi[1];
      float h = roi[4] - roi[2];
      w = std::max((float)1, w);
      h = std::max((float)1, h);
      int n_seg = poly[1];

      int offset = 2 + n_seg;
      RLE* rles;
      rlesInit(&rles, n_seg);
      for(int i = 0; i < n_seg; i++){
        int cur_len = poly[i+2];
        double* xys = new double[cur_len];
        for(int j = 0; j < cur_len; j++){
          if (j % 2 == 0)
            xys[j] = (poly[offset+j+1] - roi[2]) * mask_size / h;
          else
            xys[j] = (poly[offset+j-1] - roi[1]) * mask_size / w;


        }
        rleFrPoly(rles + i, xys, cur_len/2, mask_size, mask_size);
        delete [] xys;
        offset += cur_len;
      }
      // Decode RLE to mask
      byte* byte_mask = new byte[mask_size*mask_size*n_seg];
      rleDecode(rles, byte_mask, n_seg);
      // Flatten mask
      for(int j = 0; j < mask_size*mask_size; j++)
      {
        float cur_byte = 0;
        for(int i = 0; i< n_seg; i++){
          int offset = i * mask_size * mask_size + j;
          if(byte_mask[offset]==1){
            cur_byte = 1;
            break;
          }
        }
  mask[j] = cur_byte;
                 
      }
      
      // Check to make sure we don't have memory leak
      rlesFree(&rles, n_seg);
      delete [] byte_mask;

    }
}  // namespace utils


template<typename xpu>
class MaskRcnnTargetGPUOp : public Operator{
 public:
  float* cmask_outs, *cmask_cls;
  float* crois, *cmask_boxes, *cgt_masks, *cmask_ids;

  explicit MaskRcnnTargetGPUOp(MaskRcnnTargetParam param) {
    this->param_ = param;
    this->cmask_outs = new float[param_.batch_size*param_.num_proposals*param_.mask_size*param_.mask_size];
    this->crois = new float[param_.batch_size*param_.num_proposals*5];
    this->cgt_masks = new float[param_.batch_size*param_.max_num_gts*param_.max_polygon_len];
    this->cmask_ids = new float[param_.batch_size*param_.num_proposals];
    this->cmask_cls = new float[param_.batch_size*param_.num_proposals*param_.mask_size*param_.mask_size];
  }
  ~MaskRcnnTargetGPUOp() {
    delete [] this->cmask_outs; 
    delete [] this->crois;
    delete [] this->cgt_masks;
    delete [] this->cmask_ids;
    delete [] this->cmask_cls;
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_states) {
    CHECK_EQ(in_data.size(), 3);
    CHECK_EQ(out_data.size(), 2);
    using namespace mshadow;
    using namespace mshadow::expr;
    // The polygon format for each ground-truth object is as follows:
    // [category, num_seg, len_seg1, len_seg2,....,len_segn, seg1_x1,seg1_y1,...,seg1_xm,seg1_ym,seg2_x1,seg2_y1,...]

    // Get input
    Stream<gpu> *s = ctx.get_stream<gpu>();
    Tensor<gpu, 2> rois = in_data[mask::kRoIs].get<gpu, 2, real_t>(s);
    Tensor<gpu, 3> gt_masks = in_data[mask::kMaskPolys].get<gpu, 3, real_t>(s);\
    Tensor<gpu, 2> mask_ids = in_data[mask::kMaskIds].get<gpu, 2, real_t>(s);

    // Copy to CPU
    hipMemcpy(crois, rois.dptr_, param_.batch_size*param_.num_proposals*5*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cgt_masks, gt_masks.dptr_, param_.batch_size*param_.max_num_gts*param_.max_polygon_len*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cmask_ids, mask_ids.dptr_, param_.batch_size*param_.num_proposals*sizeof(float), hipMemcpyDeviceToHost);

    // Initialize the mask memory
    int mask_mem_size = param_.batch_size*param_.num_proposals*param_.mask_size*param_.mask_size;
    for(int i = 0; i < mask_mem_size; i++){
      cmask_outs[i] = param_.ignore_label;
    }

    // Initialize the mask classes to 0
    for(int i=0; i< mask_mem_size; i++)
      cmask_cls[i] = 0;
    // Allocate memory for binary mask
    #pragma omp parallel for num_threads(8)
    for(int i = 0; i < param_.batch_size * param_.num_proposals; i++){
        int mask_id = cmask_ids[i];
        if (mask_id == -1) {
          continue;
        }
        
        int imid = crois[5*i];
        int poly_offset = imid * param_.max_num_gts * param_.max_polygon_len + mask_id * param_.max_polygon_len; 
        // Convert the mask polygon to a binary mask
  float category = cgt_masks[poly_offset];  
        mask_utils::convertPoly2Mask(crois + i * 5, cgt_masks + poly_offset, param_.mask_size, \
         cmask_outs + i*param_.mask_size*param_.mask_size, category);
        // In our poly encoding the first element is the category

  int mask_area = param_.mask_size*param_.mask_size;
  
        for (int j = i*mask_area; j < (i+1)*mask_area; j++)
          cmask_cls[j] = category;
    }
  

    // Get output
    Stream<gpu> *so = ctx.get_stream<gpu>();    
    Tensor<gpu, 3> out_masks = out_data[mask::kMaskTargets].get<gpu, 3, real_t>(so);
    Tensor<gpu, 3> mask_cls = out_data[mask::kMaskCls].get<gpu, 3, real_t>(so);

    // Copy output to the GPU
    hipMemcpy(out_masks.dptr_, cmask_outs, \
      param_.batch_size*param_.num_proposals*param_.mask_size*param_.mask_size*sizeof(float), \
      hipMemcpyHostToDevice);
    hipMemcpy(mask_cls.dptr_, cmask_cls, \
      param_.batch_size*param_.num_proposals*param_.mask_size*param_.mask_size*sizeof(float), hipMemcpyHostToDevice);
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    CHECK_EQ(in_grad.size(), 3);

    Stream<xpu> *s = ctx.get_stream<xpu>();
    Tensor<xpu, 2> grois = in_grad[mask::kRoIs].get<xpu, 2, real_t>(s);
    Tensor<xpu, 3> gmask_polys = in_grad[mask::kMaskPolys].get<xpu, 3, real_t>(s);
    Tensor<xpu, 2> gmask_ids = in_grad[mask::kMaskIds].get<xpu, 2, real_t>(s);

    Assign(grois, req[mask::kRoIs], 0);
    Assign(gmask_polys, req[mask::kMaskPolys], 0);
    Assign(gmask_ids, req[mask::kMaskIds], 0);

  
  }

 private:
  MaskRcnnTargetParam param_;
};  // class MaskRcnnTarget

template<>
Operator *CreateOp<gpu>(MaskRcnnTargetParam param) {
  return new MaskRcnnTargetGPUOp<gpu>(param);
}

}  // namespace op
}  // namespace mxnet
